#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "optix/common/optix_params.h"
#include "optix/common/vec_math.h"
#include "optix/device/random.h"
#include "optix/device/util.h"


namespace optix {

/**
 * Launch-varying parameters.
 *
 * This params can be accessible from any module in a pipeline.
 * - declare with extern "C" and __constant__
 * - set in OptixPipelineCompileOptions
 * - filled in by optix upon optixLaunch
 */
extern "C" __constant__ LaunchParams params;


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

static __forceinline__ __device__ bool
traceOcclusion(OptixTraversableHandle handle, float3 ray_origin,
               float3 ray_direction, float tmin, float tmax) {
    unsigned int occluded = 0u;
    optixTrace(handle, ray_origin, ray_direction, tmin, tmax,
               0.0f,  // rayTime
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
               RAY_TYPE_OCCLUSION,  // SBT offset
               RAY_TYPE_COUNT,      // SBT stride
               RAY_TYPE_OCCLUSION,  // missSBTIndex
               occluded);
    return occluded;
}

//---------------------------------------------------------------------
// These program types are specified by prefixing the program’s name with the
// following
//  Ray generation          __raygen__
//  Intersection            __intersection__
//  Any-hit                 __anyhit__
//  Closest-hit             __closesthit__
//  Miss                    __miss__
//  Direct callable         __direct_callable__
//  Continuation callable   __continuation_callable__
//  Exception               __exception__
//
// Each program may call a specific set of device-side intrinsics that
// implement the actual ray-tracing-specific features
//---------------------------------------------------------------------

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__occlusion() {
    setPayloadOcclusion(true);
}

extern "C" __global__ void __closesthit__radiance() {
    const HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    const GeometryData::TriangleMesh& mesh_data =
        reinterpret_cast<const GeometryData::TriangleMesh&>(
            rt_data->geometry_data.triangle_mesh);
    const MaterialData& mat_data =
        reinterpret_cast<const MaterialData&>(rt_data->material_data);

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int prim_idx = optixGetPrimitiveIndex();
    const int3 index = mesh_data.indices[prim_idx];
    const float3 ray_dir = optixGetWorldRayDirection();
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------

    const float3 v0 = mesh_data.positions[index.x];
    const float3 v1 = mesh_data.positions[index.y];
    const float3 v2 = mesh_data.positions[index.z];
    float3 geometry_normal = normalize(cross(v1 - v0, v2 - v0));
    float3 shading_normal = geometry_normal;

    if (mesh_data.normals) {
        shading_normal = (1.f - u - v) * mesh_data.normals[index.x] +
                         u * mesh_data.normals[index.y] +
                         v * mesh_data.normals[index.z];
    }

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    geometry_normal = faceforward(geometry_normal, -ray_dir, geometry_normal);
    if (dot(geometry_normal, shading_normal) < 0.f)
        shading_normal -=
            2.f * dot(geometry_normal, shading_normal) * geometry_normal;
    shading_normal = normalize(shading_normal);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    float4 diffuseColor = mat_data.diffuse.albedo;
    if (mat_data.diffuse.albedo_tex) {
        const float2 tc = (1.f - u - v) * mesh_data.texcoords[index.x] +
                          u * mesh_data.texcoords[index.y] +
                          v * mesh_data.texcoords[index.z];

        float4 fromTexture =
            tex2D<float4>(mat_data.diffuse.albedo_tex, tc.x, tc.y);
        diffuseColor *= fromTexture;
    }

    // ------------------------------------------------------------------
    // compute shadow
    // ------------------------------------------------------------------
    const float3 surfPos = (1.f - u - v) * v0 + u * v1 + v * v2;
    // printf("%lf %lf %lf\n", surfPos.x, surfPos.y, surfPos.z);
    const float3 lightPos = make_float3(-9., 20.f, 0.f);
    const float3 lightDir = lightPos - surfPos;
    const float Ldist = length(lightPos - surfPos);

    // trace shadow ray:
    const bool occluded = traceOcclusion(
        params.handle, surfPos + 1e-3f * geometry_normal, lightDir,
        0.01f,         // tmin
        Ldist - 0.01f  // tmax
    );

    // ------------------------------------------------------------------
    // perform some simple "NdotD" shading
    // ------------------------------------------------------------------

    const float cosDN = 0.2f + .8f * fabsf(dot(ray_dir, shading_normal));
    RadiancePRD* prd = getPRD<RadiancePRD>();

    if (occluded) {
        prd->radiance = make_float3(0.f);
    }
    else {
        prd->radiance = make_float3(cosDN * diffuseColor);
    }
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

extern "C" __global__ void
__anyhit__occlusion() { /*! for this simple example, this will remain empty */
}

}  // namespace optix