#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "optix/common/optix_params.h"
#include "optix/common/vec_math.h"
#include "optix/device/random.h"
#include "optix/device/raygen.h"

namespace optix {

/**
 * Launch-varying parameters.
 *
 * This params can be accessible from any module in a pipeline.
 * - declare with extern "C" and __constant__
 * - set in OptixPipelineCompileOptions
 * - filled in by optix upon optixLaunch
 */
extern "C" __constant__ LaunchParams params;

static __forceinline__ __device__ void* unpackPointer(unsigned int i0,
                                                      unsigned int i1) {
    const unsigned long long uptr =
        static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, unsigned int& i0,
                                                   unsigned int& i1) {
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ RadiancePRD* getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

static __forceinline__ __device__ void
traceRadiance(OptixTraversableHandle handle, float3 ray_origin,
              float3 ray_direction, float tmin, float tmax, RadiancePRD* prd) {
    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(handle, ray_origin, ray_direction, tmin, tmax,
               0.0f,  // rayTime
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
               RAY_TYPE_RADIANCE,  // SBT offset
               RAY_TYPE_COUNT,     // SBT stride
               RAY_TYPE_RADIANCE,  // missSBTIndex
               u0, u1);
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    RadiancePRD* prd = getPRD();
    // set to constant white as background color
    prd->radiance = make_float3(1.f, 0.f, 0.f);
}

extern "C" __global__ void __miss__occlusion() {
    // setPayloadOcclusion(true);
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    const uint3 idx = optixGetLaunchIndex();
    float3 ray_origin, ray_direction;
    genCameraRay(params, idx, ray_origin, ray_direction);

    RadiancePRD prd;
    prd.radiance = make_float3(0.f);

    traceRadiance(params.handle, ray_origin, ray_direction,
                  0.01f,  // tmin
                  1e20f,  // tmax
                  &prd);

    const int r = int(255.99f * prd.radiance.x);
    const int g = int(255.99f * prd.radiance.y);
    const int b = int(255.99f * prd.radiance.z);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const unsigned int rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const unsigned int image_index = idx.x + idx.y * params.width;
    params.color_buffer[image_index] = rgba;
}
}  // namespace optix