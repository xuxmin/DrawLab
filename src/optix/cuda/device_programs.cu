#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "optix/device/random.h"
#include "optix/common/optix_params.h"
#include "optix/common/vec_math.h"


namespace optix {

/**
 * Launch-varying parameters.
 * 
 * This params can be accessible from any module in a pipeline.
 * - declare with extern "C" and __constant__
 * - set in OptixPipelineCompileOptions
 * - filled in by optix upon optixLaunch
*/
extern "C"  __constant__ LaunchParams params;

/**
 * The payload is associated with each ray, and is passed to all 
 * the intersection, any-hit, closest-hit and miss programs that 
 * are executed during this invocation of trace.
*/
struct RadiancePRD {
    float3       radiance;
};

static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1) {
    const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, unsigned int& i0, unsigned int& i1) {
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ RadiancePRD* getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

static __forceinline__ __device__ void
traceRadiance(OptixTraversableHandle handle, float3 ray_origin,
              float3 ray_direction, float tmin, float tmax, RadiancePRD* prd) {
    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(handle, ray_origin, ray_direction, tmin, tmax,
               0.0f,  // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT,
               RAY_TYPE_RADIANCE,  // SBT offset
               RAY_TYPE_COUNT,     // SBT stride
               RAY_TYPE_RADIANCE,  // missSBTIndex
               u0, u1);
}

static __forceinline__ __device__ bool
traceOcclusion(OptixTraversableHandle handle, float3 ray_origin,
               float3 ray_direction, float tmin, float tmax) {
    unsigned int occluded = 0u;
    optixTrace(handle, ray_origin, ray_direction, tmin, tmax,
               0.0f,  // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
               RAY_TYPE_OCCLUSION,  // SBT offset
               RAY_TYPE_COUNT,      // SBT stride
               RAY_TYPE_OCCLUSION,  // missSBTIndex
               occluded);
    return occluded;
}

//---------------------------------------------------------------------
// These program types are specified by prefixing the program’s name with the following
//  Ray generation          __raygen__ 
//  Intersection            __intersection__ 
//  Any-hit                 __anyhit__ 
//  Closest-hit             __closesthit__ 
//  Miss                    __miss__ 
//  Direct callable         __direct_callable__ 
//  Continuation callable   __continuation_callable__ 
//  Exception               __exception__
//
// Each program may call a specific set of device-side intrinsics that 
// implement the actual ray-tracing-specific features
//---------------------------------------------------------------------

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__occlusion() {
    setPayloadOcclusion(true);
}

extern "C" __global__ void __closesthit__radiance() {
    const HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    const GeometryData::TriangleMesh& mesh_data =
        reinterpret_cast<const GeometryData::TriangleMesh&>(
            rt_data->geometry_data.triangle_mesh);
    const MaterialData& mat_data =
        reinterpret_cast<const MaterialData&>(rt_data->material_data);

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int prim_idx = optixGetPrimitiveIndex();
    const int3 index  = mesh_data.indices[prim_idx];
    const float3 ray_dir = optixGetWorldRayDirection();
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------

    const float3 v0   = mesh_data.positions[index.x];
    const float3 v1   = mesh_data.positions[index.y];
    const float3 v2   = mesh_data.positions[index.z];
    float3 geometry_normal = normalize(cross(v1-v0, v2-v0));
    float3 shading_normal = geometry_normal;

    if (mesh_data.normals) {
        shading_normal = (1.f - u - v) * mesh_data.normals[index.x] +
            u * mesh_data.normals[index.y] + v * mesh_data.normals[index.z];
    }

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    geometry_normal = faceforward(geometry_normal, -ray_dir, geometry_normal);
    if (dot(geometry_normal, shading_normal) < 0.f)
        shading_normal -= 2.f*dot(geometry_normal, shading_normal) * geometry_normal;
    shading_normal = normalize(shading_normal);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    float4 diffuseColor = mat_data.diffuse.albedo;
    if (mat_data.diffuse.albedo_tex) {
        const float2 tc
            = (1.f-u-v) * mesh_data.texcoords[index.x]
            +         u * mesh_data.texcoords[index.y]
            +         v * mesh_data.texcoords[index.z];
      
        float4 fromTexture = tex2D<float4>(mat_data.diffuse.albedo_tex, tc.x, tc.y);
        diffuseColor *= fromTexture;
    }

    // ------------------------------------------------------------------
    // compute shadow
    // ------------------------------------------------------------------
    const float3 surfPos = (1.f - u - v) * v0 + u * v1 + v * v2;
    // printf("%lf %lf %lf\n", surfPos.x, surfPos.y, surfPos.z);
    const float3 lightPos = make_float3(-9., 20.f, 0.f);
    const float3 lightDir = lightPos - surfPos;
    const float  Ldist = length(lightPos - surfPos);

    // trace shadow ray:
    const bool occluded = traceOcclusion(
            params.handle,
            surfPos + 1e-3f * geometry_normal,
            lightDir,
            0.01f,         // tmin
            Ldist - 0.01f  // tmax
            );

    // ------------------------------------------------------------------
    // perform some simple "NdotD" shading
    // ------------------------------------------------------------------

    const float cosDN  = 0.2f + .8f * fabsf(dot(ray_dir, shading_normal));
    RadiancePRD* prd = getPRD();

    if (occluded) {
        prd->radiance = make_float3(0.f);
    }
    else {
        prd->radiance = make_float3(cosDN * diffuseColor);
    }
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

extern "C" __global__ void
__anyhit__occlusion() { /*! for this simple example, this will remain empty */
}


//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    RadiancePRD* prd = getPRD();
    // set to constant white as background color
    prd->radiance = make_float3(1.f, 0.f, 0.f);
}

extern "C" __global__ void __miss__occlusion() {
    // setPayloadOcclusion(true);
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    const int    w   = params.width;
    const int    h   = params.height;
    const float3 eye = params.eye;
    const float3 U   = params.U;
    const float3 V   = params.V;
    const float3 W   = params.W;

    const uint3  idx = optixGetLaunchIndex();

    const float2 d = 2.0f * make_float2(
             static_cast<float>(idx.x) / static_cast<float>(w),
             static_cast<float>(idx.y) / static_cast<float>(h)) - 1.0f;
    float3 ray_direction = normalize(d.x*U + d.y*V + W);
    float3 ray_origin    = eye;

    // printf("%lf %lf %lf\n", ray_direction.x, ray_direction.y, ray_direction.z);

    RadiancePRD prd;
    prd.radiance = make_float3(0.f);

    traceRadiance(params.handle, 
                  ray_origin,
                  ray_direction,
                  0.01f,  // tmin
                  1e20f,  // tmax
                  &prd);

    const int r = int(255.99f*prd.radiance.x);
    const int g = int(255.99f*prd.radiance.y);
    const int b = int(255.99f*prd.radiance.z);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const unsigned int rgba = 0xff000000 | (r<<0) | (g<<8) | (b<<16);

    // and write to frame buffer ...
    const unsigned int image_index = idx.x + idx.y * params.width;
    params.color_buffer[image_index] = rgba;
}
}  // namespace optix